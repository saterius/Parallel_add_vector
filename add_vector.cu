#include "hip/hip_runtime.h"
#include<stdio.h>

__global__ void parallel_vector_add(int* d_a, int* d_b, int* d_c, int* d_n) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < *d_n) {
		printf("I am anout to compute c[%d].\n", i);
		d_c[i] = d_a[i] + d_b[i];
	}
	else {
		printf("I am thread #%d, and doing nothing.\n", i);
	}
}

int main()
{
	//allocate and initialize host memory
	int n;
	scanf("%d", &n);
	int h_a[n];
	int h_b[n];
	int h_c[n];

	for (int i = 0; i < n; i++) {
		h_a[i] = i;
		h_b[i] = n - i;
	}

	//Part 1
	//allocate device memory for a, b, and c
	//copy a and b to device memory
	int *d_a, *d_b, *d_c, *d_n;
	hipMalloc((void **) &d_a, n*sizeof(int));
	hipMalloc((void **) &d_b, n*sizeof(int));
	hipMalloc((void **) &d_c, n*sizeof(int));
	hipMalloc((void **) &d_n, sizeof(int));

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMemcpy(d_a, &h_a, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &h_b, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);

	//Part 2
	//kernel launch code which let the device performs the actual vector addition
	if (n % 512)
		int amountBlock = (n/512) + 1;
	else
		int amountBlock = n/512;
	parallel_vector_add<<<amountBlock, 512>>>(d_a, d_b, d_c, d_n);
	hipDeviceSynchronize();

	//Part 3
	//copy c to host memory
	hipMemcpy(&h_c, d_c, n*sizeof(int), hipMemcpyDeviceToHost);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	//free device memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	for (int i = 0; i < n; i++) {
		printf("%d ", h_c[i]);
	}

	printf("\ntime used = %f\n", milliseconds);
}
